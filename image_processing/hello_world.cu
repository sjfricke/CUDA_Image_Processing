
#include <hip/hip_runtime.h>
#include <stdio.h>

const int N = 256;

__global__
void hello(char *a)
{
  printf("Hello from thread %d\n", threadIdx.x);
  // printf("Hello from thread %d with letter %c\n", threadIdx.x, a[threadIdx.x % 32]);
}

int main()
{
  char a[N] = "ABCDEFGHIJKLMNOPQRSTUVWXYZ01234";

  char *a_d;

  const int csize = N*sizeof(char);
  
  //printf("%s", a);

  hipMalloc( (void**)&a_d, csize );
  hipMemcpy( a_d, a, csize, hipMemcpyHostToDevice );

  hello<<<1, 256>>>(a_d);

  hipMemcpy( a, a_d, csize, hipMemcpyDeviceToHost );

  hipFree( a_d );
  
  //printf("%s\n", a);

  return EXIT_SUCCESS;
}
