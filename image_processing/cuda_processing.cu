#include "hip/hip_runtime.h"
#include "utils.h"
#include <stdio.h>

__global__ void average_linear_kernel(Pixel* image_data, Pixel* image_result, int col, int row, int image_index) {
 
  //int y = threadIdx.y+ blockIdx.y* blockDim.y;
  int x = threadIdx.x+ blockIdx.x* blockDim.x;

  if (x < (col * row)) {
    image_result[x] = image_data[x];
  }
  
  //if (y < height && x < width) {
  //  int index = width*y +x;
  //  Pixel color = image_data[index];
  // }
}

// pass in data and file width and height
// assume no alpha values in data
// returns pointer to value
Pixel* average_linear_cuda(Pixel* image_data_h, int width, int height, int image_count) {

  Pixel* image_data_d;
  Pixel* image_result_d;
  Pixel* image_result_h;
  const int image_data_size = sizeof(Pixel) * width * height;

  // allocated data to return
  hipMalloc( (void**)&image_result_d, image_data_size );
  image_result_h = (Pixel*)malloc(image_data_size);
  
  // allocated data to run on server
  hipMalloc( (void**)&image_data_d, image_data_size * image_count);
  hipMemcpy( image_data_d, image_data_h, image_data_size * image_count, hipMemcpyHostToDevice );

  dim3 dimGrid(64);
  dim3 dimBlock(32);

  printf("cuda calling\n");
  average_linear_kernel<<<dimGrid, dimBlock>>>(image_data_d, image_result_d,  width, height, image_count);
  printf("cuda called\n");
  
  hipMemcpy( image_result_h, image_result_d, image_data_size, hipMemcpyDeviceToHost );

  // checks for CUDA errors
  hipError_t errSync  = hipGetLastError();
  hipError_t errAsync = hipDeviceSynchronize();
   if (errSync != hipSuccess){ printf("Sync kernel error: %s\n", hipGetErrorString(errSync)); }
  else if (errAsync != hipSuccess) { printf("Async kernel error: %s\n", hipGetErrorString(errAsync)); }
  
  hipFree( image_data_d );
  hipFree( image_result_d);

  //  for (int i = 0; i < (width * height); i++) {
  //  printf("pixel <%d, %d> RGB: <%d, %d, %d>\n", i%width, i/width, image_result_h[i].red, image_result_h[i].green, image_result_h[i].blue);
  // }
  
  return image_result_h;
}