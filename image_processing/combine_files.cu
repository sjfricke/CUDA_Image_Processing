#include "hip/hip_runtime.h"
#include "utils.h"
#include <stdio.h>

__global__ void kernel_add_values(Pixel* image_data) {
 
  int y = threadIdx.y+ blockIdx.y* blockDim.y;
  int x = threadIdx.x+ blockIdx.x* blockDim.x;
  //if (y < height && x < width) {
  //  int index = width*y +x;
  //  Pixel color = image_data[index];
  // }
}

// pass in data and file width and height
// assume no alpha values in data
// returns pointer to value
Pixel* add_values(Pixel* image_data, int width, int height, int image_count) {

  Pixel* image_data_d;
  Pixel* image_result_d;

  const int data_size = sizeof(Pixel) * width * height;

  // allocated data to return
  hipMalloc( (void**)&image_result_d, data_size );

  // allocated data to run on serve
  hipMalloc( (void**)&image_data_d, data_size * image_count);
  hipMemcpy( image_data_d, image_data, data_size * image_count, hipMemcpyHostToDevice );

  dim3 dimGrid(32);
  dim3 dimBlock(32,32);
  
  kernel_add_values<<<dimGrid, dimBlock>>>(image_data_d);

  hipMemcpy( image_data, image_data_d, data_size, hipMemcpyDeviceToHost );

  hipFree( image_data_d );

  return image_result_d;
  // hipDeviceSynchronize();

  //TODO, what is this?
  //checkCudaErrors(hipGetLastError());
}