#include "hip/hip_runtime.h"
#include "utils.h"
#include <stdio.h>

__global__ void average_linear_kernel(Pixel* image_data, int col, int row, int image_index) {
 
  int y = threadIdx.y+ blockIdx.y* blockDim.y;
  int x = threadIdx.x+ blockIdx.x* blockDim.x;
  //if (y < height && x < width) {
  //  int index = width*y +x;
  //  Pixel color = image_data[index];
  // }
}

// pass in data and file width and height
// assume no alpha values in data
// returns pointer to value
Pixel* average_linear_cuda(Pixel* image_data, int width, int height, int image_count) {

  Pixel* image_data_d;
  Pixel* image_result_d;

  const int image_data_size = sizeof(Pixel) * width * height;

  // allocated data to return
  hipMalloc( (void**)&image_result_d, image_data_size );

  // allocated data to run on server
  hipMalloc( (void**)&image_data_d, image_data_size * image_count);
  hipMemcpy( image_data_d, image_data, image_data_size * image_count, hipMemcpyHostToDevice );

  dim3 dimGrid(32);
  dim3 dimBlock(32,32);
  
  average_linear_kernel<<<dimGrid, dimBlock>>>(image_data_d, width, height, image_count);

  hipMemcpy( image_data, image_data_d, image_data_size, hipMemcpyDeviceToHost );

  // checks for CUDA errors
  hipError_t errSync  = hipGetLastError();
  hipError_t errAsync = hipDeviceSynchronize();
  if (errSync != hipSuccess){ printf("Sync kernel error: %s\n", hipGetErrorString(errSync)); }
  else if (errAsync != hipSuccess) { printf("Async kernel error: %s\n", hipGetErrorString(errAsync)); }
  
  hipFree( image_data_d );
  
  return image_result_d;
}