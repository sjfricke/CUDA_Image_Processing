#include "hip/hip_runtime.h"
#include "utils.h"
#include <stdio.h>

__global__ void kernel_add_values(Pixel* image_data, Pixel* output_data, int width, int height) {
 
  int y = threadIdx.y+ blockIdx.y* blockDim.y;
  int x = threadIdx.x+ blockIdx.x* blockDim.x;
  if (y < height && x < width) {
    int index = width*y +x;
    Pixel color = image_data[index];
  }
}

// pass in data and file width and height
// assume no alpha values in data
// returns pointer to value
Pixel* add_values(Pixel* image_data, int width, int height, int image_count) {

  Pixel* image_data_d;
  Pixel* image_result_d;

  const int file_data_size = sizeof(Pixel) * width * height;

  // allocated data to return
  hipMalloc( (void**)&image_result_d, data_size );

  // allocated data to run on serve
  hipMalloc( (void**)&image_data_d, data_size * image_count);
  hipMemcpy( image_data_d, image_data, data_size * image_count, hipMemcpyHostToDevice );

  dim3 dimGrid(32);
  dim3 dimBlock(32,32);
  
  add_values<<<dimGrid, dimBlock>>>(image_data_d, image_results_d, width, height);

  hipMemcpy( image_data, image_data_d, csize, hipMemcpyDeviceToHost );

  hipFree( image_data_d );
  
  // hipDeviceSynchronize();

  //TODO, what is this?
  //checkCudaErrors(hipGetLastError());
}