#include "hip/hip_runtime.h"
#include "utils.h"
#include <stdio.h>

// HARD coding 128 images into it for the time being
// Parallel Reduction is not a fun task to work with...
// unrolling loop to save on preformance as much as possible
// by unrolling it the kernel gets evaluated at compile time for the if statments
__global__ void average_linear_kernel(Pixel* image_data, Pixel* image_result, int image_size) {
   
  __shared__ unsigned int pixel_reduction[64 * 3]; // only need first half of shared array

  // int y = threadIdx.y+ blockIdx.y* blockDim.y;
  // int x = threadIdx.x+ blockIdx.x* blockDim.x;
  int t_id = threadIdx.x; // which image index it is
  int block_offset = blockIdx.x * image_size;
  
  if (t_id < 64) {
    pixel_reduction[(t_id*3) + 0] = image_data[t_id + block_offset].red   + image_data[t_id + block_offset + 64].red;
    pixel_reduction[(t_id*3) + 1] = image_data[t_id + block_offset].green + image_data[t_id + block_offset + 64].green;
    pixel_reduction[(t_id*3) + 2] = image_data[t_id + block_offset].blue  + image_data[t_id + block_offset + 64].blue;
  }
  __syncthreads();

  // Instructions are SIMD synchronous within a warp
  if (t_id < 32) {
    pixel_reduction[(t_id*3) + 0] += pixel_reduction[((t_id+32)*3) + 0];
    pixel_reduction[(t_id*3) + 1] += pixel_reduction[((t_id+32)*3) + 1];
    pixel_reduction[(t_id*3) + 2] += pixel_reduction[((t_id+32)*3) + 2]; 
  __syncthreads();

    pixel_reduction[(t_id*3) + 0] += pixel_reduction[((t_id+16)*3) + 0];
    pixel_reduction[(t_id*3) + 1] += pixel_reduction[((t_id+16)*3) + 1];
    pixel_reduction[(t_id*3) + 2] += pixel_reduction[((t_id+16)*3) + 2];
  __syncthreads();

    pixel_reduction[(t_id*3) + 0] += pixel_reduction[((t_id+8)*3) + 0];
    pixel_reduction[(t_id*3) + 1] += pixel_reduction[((t_id+8)*3) + 1];
    pixel_reduction[(t_id*3) + 2] += pixel_reduction[((t_id+8)*3) + 2];
  __syncthreads();

    pixel_reduction[(t_id*3) + 0] += pixel_reduction[((t_id+4)*3) + 0];
    pixel_reduction[(t_id*3) + 1] += pixel_reduction[((t_id+4)*3) + 1];
    pixel_reduction[(t_id*3) + 2] += pixel_reduction[((t_id+4)*3) + 2];
  __syncthreads();

    pixel_reduction[(t_id*3) + 0] += pixel_reduction[((t_id+2)*3) + 0];
    pixel_reduction[(t_id*3) + 1] += pixel_reduction[((t_id+2)*3) + 1];
    pixel_reduction[(t_id*3) + 2] += pixel_reduction[((t_id+2)*3) + 2];
  __syncthreads();

    pixel_reduction[(t_id*3) + 0] += pixel_reduction[((t_id+1)*3) + 0];
    pixel_reduction[(t_id*3) + 1] += pixel_reduction[((t_id+1)*3) + 1];
    pixel_reduction[(t_id*3) + 2] += pixel_reduction[((t_id+1)*3) + 2];
  }
  __syncthreads();

  if (t_id == 0) {
    image_result[blockIdx.x].red   = pixel_reduction[0];
    image_result[blockIdx.x].green = pixel_reduction[1];
    image_result[blockIdx.x].blue  = pixel_reduction[2];
  }
  __syncthreads();

}

// pass in data and file width and height
// assume no alpha values in data
// returns pointer to value
Pixel* average_linear_cuda(Pixel* image_data_h, int width, int height, int image_count) {

  Pixel* image_data_d;
  Pixel* image_result_d;
  Pixel* image_result_h;
  const int image_size = width * height;
  const int image_data_size = sizeof(Pixel) * image_size;
  
  // allocated data to return
  hipMalloc( (void**)&image_result_d, image_data_size );
  image_result_h = (Pixel*)malloc(image_data_size);
  
  // allocated data to run on server
  hipMalloc( (void**)&image_data_d, image_data_size * image_count);
  hipMemcpy( image_data_d, image_data_h, image_data_size * image_count, hipMemcpyHostToDevice );

  dim3 dimGrid(height * width);
  dim3 dimBlock(128); //HARD set atm

  printf("cuda calling\n");
  average_linear_kernel<<<dimGrid, dimBlock>>>(image_data_d, image_result_d, image_size);
  printf("cuda called\n");
  
  hipMemcpy( image_result_h, image_result_d, image_data_size, hipMemcpyDeviceToHost );

  // checks for CUDA errors
  hipError_t errSync  = hipGetLastError();
  hipError_t errAsync = hipDeviceSynchronize();
   if (errSync != hipSuccess){ printf("Sync kernel error: %s\n", hipGetErrorString(errSync)); }
  else if (errAsync != hipSuccess) { printf("Async kernel error: %s\n", hipGetErrorString(errAsync)); }
  
  hipFree( image_data_d );
  hipFree( image_result_d);

  //  for (int i = 0; i < (width * height); i++) {
  //  printf("pixel <%d, %d> RGB: <%d, %d, %d>\n", i%width, i/width, image_result_h[i].red, image_result_h[i].green, image_result_h[i].blue);
  // }
  
  return image_result_h;
}